#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>

// Kernel to compute force matrix


__global__ void computeForces(double* forceX, double* forceY, double* xPos, double* yPos,
    int N, int A, int B, double* sigma, const double epsilon)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N)
    {
        double dx = xPos[j] - xPos[i];
        double dy = yPos[j] - yPos[i];
        double sep = sqrt(dx * dx + dy * dy);

        if (sep > 1e-9) { // Small epsilon to avoid division by zero
            double invr7 = 1.0 / (sep * sep * sep * sep * sep * sep * sep);
            double invr13 = invr7 / (sep * sep * sep * sep * sep * sep);

            double force = 4 * epsilon * ((A * pow(sigma[i], 6)) * invr7 - (B * pow(sigma[i], 12)) * invr13);
            forceX[i * N + j] = (dx / sep) * force;
            forceY[i * N + j] = (dy / sep) * force;
        }
        else {
            forceX[i * N + j] = 0.0;
            forceY[i * N + j] = 0.0;
        }
    }

}

// Kernel to aggregate accelerations
__global__ void aggregateAccelerations(double* forceX, double* forceY, double* accX, double* accY, const double* masses, int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N)
    {
        double sumX = 0;
        double sumY = 0;
        for (int col = 0; col < N; col++)
        {
            sumX += forceX[row * N + col];
            sumY += forceY[row * N + col];
        }
        accX[row] = sumX / masses[row];
        accY[row] = sumY / masses[row];

    }
}


// verlet integration kernel
__global__ void integratePositions(int count, double* dev_xPosMatrix, double* dev_yPosMatrix, double* xPos, double* yPos,
    double* xVel, double* yVel, double* accX, double* accY, int N, double timeStep, double* radii, double bw) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int sample = 100;
    if (i < N) {
        if (count % sample == 0) {
            int row = count / sample;
            dev_xPosMatrix[row * N + i] = xPos[i];
            dev_yPosMatrix[row * N + i] = yPos[i];
        }
        xPos[i] += xVel[i] * timeStep + 0.5 * accX[i] * timeStep * timeStep;
        yPos[i] += yVel[i] * timeStep + 0.5 * accY[i] * timeStep * timeStep;

        // Handle boundary conditions after position update
        if (((xPos[i] - radii[i]) <= 0) || ((xPos[i] + radii[i]) >= bw)) {
            xVel[i] = -xVel[i];
        }
        if (((yPos[i] - radii[i]) <= 0) || ((yPos[i] + radii[i]) >= bw)) {
            yVel[i] = -yVel[i];
        }
    }
}

// verlet velocity integration kernel
__global__ void integrateVelocities(double* xVel, double* yVel, double* oldAccX, double* oldAccY, double* newAccX, double* newAccY, int N, double timeStep) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        xVel[i] += 0.5 * (oldAccX[i] + newAccX[i]) * timeStep;
        yVel[i] += 0.5 * (oldAccY[i] + newAccY[i]) * timeStep;  // Use newAccY here

        oldAccX[i] = newAccX[i];
        oldAccY[i] = newAccY[i];
    }
}


// compute accelerations
hipError_t computeAccelerations(double* dev_forceX, double* dev_forceY, double* dev_xPos, double* dev_yPos, double* dev_masses,
    double* dev_accX, double* dev_accY, double* dev_sigma,
    const unsigned int N, const double A, const double B, const double epsilon, const double timeStep)
{

    int threadsPerBlock = 16;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    dim3 dimBlock(threadsPerBlock, threadsPerBlock);
    dim3 dimGrid(blocksPerGrid, blocksPerGrid);

    computeForces << <dimGrid, dimBlock >> > (dev_forceX, dev_forceY, dev_xPos, dev_yPos,
        N, A, B, dev_sigma, epsilon);
    hipError_t cudaStatus = hipDeviceSynchronize();
    // Aggregate accelerations
    // Using a 1D grid since we're aggregating over rows
    threadsPerBlock = 128;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    aggregateAccelerations << <blocksPerGrid, threadsPerBlock >> > (dev_forceX, dev_forceY, dev_accX, dev_accY, dev_masses, N);
    cudaStatus = hipDeviceSynchronize();

    return cudaStatus;
}

// Function to write a matrix to a CSV file
void writeMatrixToFile(double* matrix, int rows, int cols, const char* filename) {
    FILE* file = fopen(filename, "w");
    if (file == NULL) {
        fprintf(stderr, "Error opening file %s for writing.\n", filename);
        return;
    }
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(file, "%f", matrix[i * cols + j]);
            if (j < cols - 1) fprintf(file, ","); // No trailing comma at the end of the row
        }
        fprintf(file, "\n"); // New line at the end of each row
    }
    fclose(file);
}


void printMatrix(double* matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}





int main()
{
    const double timeStep = 1e-4;
    const int runTime = 10;
    int samplerate = 100;
    const int N = 450;
    const double epsilon = 0.1;
    const double A = 0.4;
    const double B = 1;
    double speed = 7;
    const int iterations = runTime / timeStep;
    const double boxwidth = 37.0;

    // Allocate memory
    double* xPos = (double*)malloc(N * sizeof(double));
    double* yPos = (double*)malloc(N * sizeof(double));
    double* xVel = (double*)malloc(N * sizeof(double));
    double* yVel = (double*)malloc(N * sizeof(double));
    double* masses = (double*)malloc(N * sizeof(double));
    double* sigma = (double*)malloc(N * sizeof(double));
    double* radii = (double*)malloc(N * sizeof(double));
    double* xPositionMatrix = (double*)malloc((iterations / samplerate) * N * sizeof(double));
    double* yPositionMatrix = (double*)malloc((iterations / samplerate) * N * sizeof(double));

    // Initialize positions, velocities, etc.
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        masses[i] = 1;
        xPos[i] = (double)rand() / (double)(RAND_MAX / (boxwidth - 1));
        yPos[i] = (double)rand() / (double)(RAND_MAX / (boxwidth - 1));
        xVel[i] = ((double)rand() / (double)(RAND_MAX / speed)) - (speed / 2);
        yVel[i] = ((double)rand() / (double)(RAND_MAX / speed)) - (speed / 2);
        radii[i] = 0.7;
        sigma[i] = 0.7 / pow(2, 1 / 6);
    }
    masses[N / 2] = 4.5e10; // Brownian particle
    radii[N / 2] = 1.4;
    xVel[N / 2] = 0;
    yVel[N / 2] = 0;

    // Allocate device memory
    double* dev_xPos, * dev_yPos, * dev_xVel, * dev_yVel, * dev_accX, * dev_accY;
    double* dev_newaccX, * dev_newaccY, * dev_sigma, * dev_masses, * dev_radii;
    double* dev_xmat, * dev_ymat;

    hipMalloc((void**)&dev_xPos, N * sizeof(double));
    hipMalloc((void**)&dev_yPos, N * sizeof(double));
    hipMalloc((void**)&dev_xVel, N * sizeof(double));
    hipMalloc((void**)&dev_yVel, N * sizeof(double));
    hipMalloc((void**)&dev_accX, N * sizeof(double));
    hipMalloc((void**)&dev_accY, N * sizeof(double));
    hipMalloc((void**)&dev_newaccX, N * sizeof(double));
    hipMalloc((void**)&dev_newaccY, N * sizeof(double));
    hipMalloc((void**)&dev_sigma, N * sizeof(double));
    hipMalloc((void**)&dev_masses, N * sizeof(double));
    hipMalloc((void**)&dev_radii, N * sizeof(double));
    hipMalloc((void**)&dev_xmat, (iterations / samplerate) * N * sizeof(double));
    hipMalloc((void**)&dev_ymat, (iterations / samplerate) * N * sizeof(double));

    double* dev_forceX;
    double* dev_forceY;
    hipMalloc((void**)&dev_forceX, N * N * sizeof(double));
    hipMalloc((void**)&dev_forceY, N * N * sizeof(double));

    // Copy data to device
    hipMemcpy(dev_xPos, xPos, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_yPos, yPos, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_xVel, xVel, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_yVel, yVel, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_sigma, sigma, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_masses, masses, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_radii, radii, N * sizeof(double), hipMemcpyHostToDevice);

    // Compute initial accelerations
    computeAccelerations(dev_forceX, dev_forceY, dev_xPos, dev_yPos, dev_masses, dev_accX, dev_accY, dev_sigma, N, A, B, epsilon, timeStep);

    int threadsPerBlock = 128;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Main loop
    for (int count = 0; count < iterations; count++) {
        integratePositions << <blocksPerGrid, threadsPerBlock >> > (count, dev_xmat, dev_ymat, dev_xPos, dev_yPos,
            dev_xVel, dev_yVel, dev_accX, dev_accY, N, timeStep, dev_radii, boxwidth);

        hipDeviceSynchronize();

        // Compute new accelerations after positions are updated
        computeAccelerations(dev_forceX, dev_forceY, dev_xPos, dev_yPos, dev_masses, dev_newaccX, dev_newaccY,
            dev_sigma, N, A, B, epsilon, timeStep);

        // Update velocities using old and new accelerations
        integrateVelocities << <blocksPerGrid, threadsPerBlock >> > (dev_xVel, dev_yVel, dev_accX, dev_accY,
            dev_newaccX, dev_newaccY, N, timeStep);

        hipDeviceSynchronize();

    }

    // Copy results back to host
    hipMemcpy(xPositionMatrix, dev_xmat, (iterations / samplerate) * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(yPositionMatrix, dev_ymat, (iterations / samplerate) * N * sizeof(double), hipMemcpyDeviceToHost);

    writeMatrixToFile(xPositionMatrix, iterations / samplerate, N, "xPositionMatrix.csv");
    writeMatrixToFile(yPositionMatrix, iterations / samplerate, N, "yPositionMatrix.csv");


    // Free memory
    free(xPos);
    free(yPos);
    free(xVel);
    free(yVel);
    free(masses);
    free(sigma);
    free(radii);
    free(xPositionMatrix);
    free(yPositionMatrix);

    hipFree(dev_xPos);
    hipFree(dev_yPos);
    hipFree(dev_xVel);
    hipFree(dev_yVel);
    hipFree(dev_accX);
    hipFree(dev_accY);
    hipFree(dev_newaccX);
    hipFree(dev_newaccY);
    hipFree(dev_sigma);
    hipFree(dev_masses);
    hipFree(dev_radii);
    hipFree(dev_xmat);
    hipFree(dev_ymat);
    hipFree(dev_forceX);
    hipFree(dev_forceY);

    return 0;
}
